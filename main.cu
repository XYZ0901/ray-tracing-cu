#include "hip/hip_runtime.h"
//
// Created by root on 1/28/23.
//

#include <iostream>
#include <cfloat>

#include <hiprand/hiprand_kernel.h>

#include "heads/rtweekend.h"

#include "heads/global.h"
#include "heads/vec3.h"
#include "heads/color.h"
#include "heads/ray.h"
#include "heads/hittable.h"
#include "heads/sphere.h"
#include "heads/hittable_list.h"
#include "heads/camera.h"
#include "heads/material.h"

#define sample_per_pixel_float 320.f
#define sample_per_pixel_int 320
#define _aspect_ratio (3.f/2.f)
#define _R cos(pi/4.f)

__device__ color ray_color(const ray &r, hittable **world, hiprandState *st) {
    ray cur_ray = r;
    vec3 cur_attenuation{1., 1., 1.};

    for (int i = 0; i < 50; i++) {
        hit_record rec;
        if ((*world)->hit(cur_ray, 0.1f, FLT_MAX, rec)) {
            ray scattered;
            vec3 attenuation;
            if (rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, st)) {
                cur_attenuation = cur_attenuation * attenuation;
                cur_ray = scattered;
            } else {
                return {0, 0, 0};
            }
        } else {
            vec3 unit_direction = unit_vector(cur_ray.direction());
            auto t = 0.5f * (unit_direction.y() + 1.f);
            vec3 c = (1.f - t) * color(1.f, 1.f, 1.f) + t * color(.5f, .7f, 1.f);
            return cur_attenuation * c;
        }
    }
    return {0, 0, 0};
}

__global__ void render(vec3 *fb, camera **cam, hittable **world) {
    int i = blockIdx.x;
    int j = blockIdx.y;
    int t = threadIdx.x;

    __shared__ float cache[sample_per_pixel_int * 3];

    int max_x = gridDim.x;
    int max_y = gridDim.y;

    int pixel_index = (j * max_x + i);

    hiprandState st;
    hiprand_init(pixel_index * sample_per_pixel_float + t, 0, 0, &st);

    float u = float(i + hiprand_uniform(&st)) / float(max_x - 1);
    float v = float(j + hiprand_uniform(&st)) / float(max_y - 1);

    ray r = (*cam)->get_ray(u, v, &st);
    color pixel_color = ray_color(r, world, &st);

    cache[t * 3 + 0] = pixel_color.x();
    cache[t * 3 + 1] = pixel_color.y();
    cache[t * 3 + 2] = pixel_color.z();

    __syncthreads();

    int idx = blockDim.x >> 1;
    while (idx != 0) {
        if (t < idx) {
            cache[t * 3 + 0] += cache[(t + idx) * 3 + 0];
            cache[t * 3 + 1] += cache[(t + idx) * 3 + 1];
            cache[t * 3 + 2] += cache[(t + idx) * 3 + 2];
        }
        __syncthreads();
        idx >>= 1;
    }

    if (t == 0) {
        color pixel = color{cache[0],
                            cache[1],
                            cache[2]} / sample_per_pixel_float;
        fb[pixel_index] = pixel;
    }
}

#define RND (hiprand_uniform(&st))

//__global__ void create_world(hittable **d_list, hittable **d_world, camera **cam) {
//    if (threadIdx.x == 0 && blockIdx.x == 0) {
//        hiprandState st;
//        hiprand_init(1234, 0, 0, &st);
//        auto ground_material = new lambertian({.5, .5, .5});
//        d_list[0] = new sphere({0, -1000, 0}, 1000, ground_material);
////        int i = 1;
//        for (int a = -11; a < 11; a++) {
//            for (int b = -11; b < 11; b++) {
//                auto choose_mat = RND;
//                point3 center(a + .9 * RND, .2, b + .9 * RND);
//                int idx = ((a + 11) * 22 + b + 11) + 1;
//                d_list[idx] = nullptr;
//                if ((center - point3(4, .2, 0)).length() > .9) {
//                    material *sphere_material;
//                    if (choose_mat < .8) {
//                        auto albedo = _RANDVEC3 * _RANDVEC3;
//                        sphere_material = new lambertian(albedo);
//                    } else if (choose_mat < .95) {
//                        auto albedo = _RANDVEC3 / 2 + color{.5, .5, .5};
//                        auto fuzz = hiprand_uniform(&st) / 2;
//                        sphere_material = new metal(albedo, fuzz);
//                    } else {
//                        sphere_material = new dielectric(1.5);
//                    }
//                    d_list[idx] = new sphere(center, .2, sphere_material);
//                }
//            }
//        }
//
//        d_list[484 + 1] = new sphere({0, 1, 0}, 1, new dielectric(1.5));
//        d_list[484 + 2] = new sphere({-4, 1, 0}, 1, new lambertian({.4, .2, .1}));
//        d_list[484 + 3] = new sphere({4, 1, 0}, 1, new metal({.7, .6, .5}, 0.));
//
//        *d_world = new hittable_list(d_list, (484 + 1 + 3));
//
//        point3 lookfrom(13, 2, 3);
//        point3 lookat(0, 0, 0);
//        vec3 vup(0, 1, 0);
//        auto dist_to_focus = 10.f;
//        auto aperture = .1f;
//
//        *cam = new camera(lookfrom, lookat,
//                          vup, 20, _aspect_ratio, aperture, dist_to_focus);
//    }
//}

__global__ void create_world(hittable **d_list, hittable **d_world, camera **cam) {

    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprandState st;
        hiprand_init(0, 0, 0, &st);

        d_list[0] = new sphere(vec3(0, -1000.0, -1), 1000,
                               new lambertian(vec3(0.5, 0.5, 0.5)));

        int i = 1;
        for (int a = -11; a < 11; a++) {
            for (int b = -11; b < 11; b++) {
                float choose_mat = RND;
                vec3 center(a + RND, 0.2, b + RND);
                if (choose_mat < 0.8f) {
                    d_list[i++] = new sphere(center, 0.2,
                                             new lambertian(vec3(RND * RND, RND * RND, RND * RND)));
                } else if (choose_mat < 0.95f) {
                    d_list[i++] = new sphere(center, 0.2,
                                             new metal(vec3(0.5f * (1.0f + RND), 0.5f * (1.0f + RND),
                                                            0.5f * (1.0f + RND)), 0.5f * RND));
                } else {
                    d_list[i++] = new sphere(center, 0.2, new dielectric(1.5));
                }
            }
        }

        d_list[i++] = new sphere(vec3(0, 1, 0), 1.0, new dielectric(1.5));
        d_list[i++] = new sphere(vec3(-4, 1, 0), 1.0, new lambertian(vec3(0.4, 0.2, 0.1)));
        d_list[i++] = new sphere(vec3(4, 1, 0), 1.0, new metal(vec3(0.7, 0.6, 0.5), 0.0));

        *d_world = new hittable_list(d_list, 22 * 22 + 1 + 3);

        vec3 lookfrom(13, 2, 3);
        vec3 lookat(0, 0, 0);
        vec3 vup{0, 1, 0};
        float dist_to_focus = 10.0;
        float aperture = 0.;
        *cam = new camera(lookfrom,
                          lookat,
                          vup,
                          20.0,
                          _aspect_ratio,
                          aperture,
                          dist_to_focus);
    }

}

__global__ void free_world(hittable **d_list, hittable **d_world, camera **cam) {
    for (int i = 0; i < 22*22+1+3; i++) {
//        if (i != 3)
//        if (d_list[i] == nullptr) {
//            continue;
//        }
        delete ((sphere *) d_list[i])->mat_ptr;
        delete d_list[i];
    }

    delete *d_world;
    delete *cam;
}

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state) {
    int i = blockIdx.x;
    int j = blockIdx.y;

    int pixel_idx = j * max_x + i;

    hiprand_init(1999, pixel_idx, 0, &rand_state[pixel_idx]);
}

int main(int argc, char **argv) {
    // Image
    const auto aspect_ratio = _aspect_ratio;
    const int image_width = 1200;
    const int image_height = static_cast<int>(image_width / aspect_ratio);

    int num_pixels = image_height * image_width;
    size_t fb_size = num_pixels * sizeof(vec3);

    // allocate FB
    vec3 *fb;
    checkCudaErrors(hipMalloc((void **) &fb, fb_size));

    // World
    hittable **d_list;
    int num_hitables = 22 * 22 + 1 + 3;
    checkCudaErrors(hipMalloc((void **) &d_list, num_hitables * sizeof(hittable *)));
    hittable **d_world;
    checkCudaErrors(hipMalloc((void **) &d_world, sizeof(hittable *)));
    // Camera
    camera **d_cam;
    checkCudaErrors(hipMalloc((void **) &d_cam, sizeof(camera *)));
    create_world<<<1, 1>>>(d_list, d_world, d_cam);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    dim3 block_size(image_width, image_height);
    dim3 thread_size(sample_per_pixel_int);

//    hiprandState *d_rand_state;
//    checkCudaErrors(hipMalloc((void **) &d_rand_state, num_pixels * sizeof(hiprandState)));
//    render_init<<<block_size, 1>>>(image_width, image_height, d_rand_state);
//    checkCudaErrors(hipGetLastError());
//    checkCudaErrors(hipDeviceSynchronize());

    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    checkCudaErrors(hipEventRecord(start));

    render<<<block_size, thread_size>>>(fb, d_cam, d_world);

    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipEventSynchronize(stop));
    float elapsed_time;
    checkCudaErrors(hipEventElapsedTime(&elapsed_time, start, stop));
    printf("Time = %g ms.\n", elapsed_time);
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

    free_world<<<1, 1>>>(d_list, d_world, d_cam);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_cam));

    auto rb = (vec3 *) calloc(num_pixels, sizeof(vec3));
    checkCudaErrors(hipMemcpy(rb, fb, num_pixels * sizeof(vec3), hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(fb));

    // Render
//    std::cout << "P3\n" << image_width << ' ' << image_height << "\n255\n";

    FILE *fp = fopen("./image.ppm", "wb");
    (void) fprintf(fp, "P6\n%d %d\n255\n", image_width, image_height);
    for (int j = image_height - 1; j >= 0; --j) {
//        std::cerr << "\rScanlines remaining: " << j << ' ';
//        std::cerr.flush();
        for (int i = 0; i < image_width; ++i) {
            size_t pixel_idx = (j * image_width + i);
            color pixel_color = rb[pixel_idx];
            pixel_color.e[0] = sqrt(pixel_color.x());
            pixel_color.e[1] = sqrt(pixel_color.y());
            pixel_color.e[2] = sqrt(pixel_color.z());
            write_color(fp, pixel_color);
//            write_color(std::cout, pixel_color);
        }
        UpdateProgress(j / (float) image_height);
    }
    fclose(fp);

    return 0;
}
